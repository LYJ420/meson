#include <hip/hip_runtime.h>
#include <iostream>

int cuda_devices(void) {
    int result = 0;
    hipGetDeviceCount(&result);
    return result;
}


int main(void) {
    int n = cuda_devices();
    if (n == 0) {
        std::cout << "No Cuda hardware found. Exiting.\n";
        return 0;
    }

    std::cout << "Found " << n << "Cuda devices.\n";
    return 0;
}
